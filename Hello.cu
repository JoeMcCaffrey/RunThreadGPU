#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

/*

This program moves data from main memory to the GPU memory and prints
using threads by exploiting the CPU's synconization
*/

// Compile: nvcc Hello.cu -o Hello



// this is the global gpu function that will run on the threads
__global__ void hello(float *a, float *r) {
  int thread=blockIdx.x * blockDim.x + threadIdx.x;

  r[thread]=a[thread];

  printf("TID=%d:  Hello World! a=%f\n", thread, a[thread]);
}

main() {
// we have some data stored in cpu memory, adata, bdata,rdata
// and two defined variables that will live in gpu data
   float aData[32], bData[32], rData[32], *aGPU, *rGPU;
   int index;

// we init the array that is in cpu memory
   for(index=0;index<32;index++) {
     aData[index]=5*index;
   }
// we init the gpu memory of some size
   hipMalloc((void **)&aGPU, sizeof(float)*32);
   hipMalloc((void **)&rGPU, sizeof(float)*32);
// copy the memory from cpu land to gpu land so we can run it on the gpu
   hipMemcpy(aGPU, aData, sizeof(float)*32, hipMemcpyHostToDevice);
// this is the call to threads to run on the gpu, see we have defined the agpu and rgpu
// we want to call the cuda device sycronize to wait for the threads to execute
   hello<<<1, 32>>>(aGPU, rGPU);
// the above have to equal the size of the array (multiplication) 

   hipDeviceSynchronize();
// we want to copy the results back from gpu land to cpu land
   hipMemcpy(rData, rGPU, sizeof(float)*32, hipMemcpyDeviceToHost);
   int i ;
// print the results 
   for(i = 0; i < 32 ; i++){
    printf("result: r[%d]=%f\n",i, rData[i]);
   }
}


